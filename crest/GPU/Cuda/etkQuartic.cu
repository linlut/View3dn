#include "hip/hip_runtime.h"
//FILE: etkQuartic.cu

#ifndef etkQuadric_hpp
#define etkQuadric_hpp
struct QuarticSolverResult
{
	int num;			//number of roots
	QFLOAT roots[4];	//the four roots
};
#endif

//=====================Local definitions=====================
#define doub0 0.0
#define doub1 1.0
#define doub2 2.0
#define doub3 3.0
#define doub4 4.0
#define doub6 6.0
#define doub12 12
#define doub24 24
#define inv2 0.5
#define inv3 0.3333333333333333333333333
#define inv4 0.250

#define doubmax 1e36	//Approx square root of max QFLOAT number
#define doubtol 1e-6	//Tolerance for deciding the quadirc function or bi-quadirc function
						//This value seems to be very important for single precision computation
						//For double precision computation, it is not sensitive to this value
//=====================Local definitions=====================
//Define sqrt(3.0);
#define rt3 1.7320508075688772935274463415059


// ----------------------------------------------------------------------------
// find cos(acos(x)/3)
// Don Herbison-Evans   16/7/81
// called by cubic .

__device__ inline QFLOAT acos3(const QFLOAT x)
{
   const QFLOAT value= cos(acos(x)*inv3);
   return value;
}

// ----------------------------------------------------------------------------
// find cube root of x.
// Don Herbison-Evans   30/1/89
// called by cubic .

__device__ inline QFLOAT curoot(const QFLOAT x)
{
   QFLOAT value;
   QFLOAT absx;
   int neg;

   neg = 0;
   absx = x;
   if (x < doub0){
      absx = -x;
      neg = 1;
   }
   if (absx != doub0) 
	   value = exp( log(absx)*inv3 );
   else 
	   value = doub0;
   if (neg == 1) 
	   value = -value;
   return(value);
}

// ----------------------------------------------------------------------------
// find the errors
// called by quartic.
/*
__device__ void errors(const QFLOAT a, const QFLOAT b, const QFLOAT c, const QFLOAT d, QFLOAT rts[4], QFLOAT rterr[4], int nrts)
{
   int k;
   QFLOAT deriv, test;

   if (nrts > 0){
      for (k = 0 ; k < nrts ; ++ k){
         test = (((rts[k]+a)*rts[k]+b)*rts[k]+c)*rts[k]+d ;
         if (test == doub0) 
			 rterr[k] = doub0;
         else{
            deriv =((doub4*rts[k]+doub3*a)*rts[k]+doub2*b)*rts[k]+c ;
            if (deriv != doub0)
               rterr[k] = fabs(test/deriv);
            else{
               deriv = (doub12*rts[k]+doub6*a)*rts[k]+doub2*b ;
               if (deriv != doub0)
                   rterr[k] = sqrt(fabs(test/deriv)) ;
               else{
                  deriv = doub24*rts[k]+doub6*a ;
                  if (deriv != doub0)
                     rterr[k] = curoot(fabs(test/deriv));
                  else
                     rterr[k] = sqrt(sqrt(fabs(test)/doub24));
               }
            }
         }
         if (rts[k] != doub0) 
			 rterr[k] /= rts[k];
         if (rterr[k] < doub0) 
			 rterr[k] = -rterr[k];
      }
   }
}
*/

// ----------------------------------------------------------------------------
// solve the quadratic equation -
//  x**2+b*x+c = 0
// called by  quartic, descartes, ferrari, neumark.
__device__ inline QuarticSolverResult 
quadratic(const QFLOAT b, const QFLOAT c, const QFLOAT dis)
{
	QuarticSolverResult result;

	if (dis > doub0){
		QFLOAT rts[2];
		QFLOAT rtdis = sqrt(dis) ;
		if (b > doub0) 
			rts[0] = (-b - rtdis)*inv2 ;
		else 
			rts[0] = (-b + rtdis)*inv2 ;
		if (rts[0] == doub0) 
			rts[1] = -b ;
		else 
			rts[1] = c/rts[0] ;
		result.num = 2 ;
		result.roots[0] = rts[0];
		result.roots[1] = rts[1];
	}
	else{
		result.num = 0;
		result.roots[0] = result.roots[1] = 0;
	}

	return result;
}

/*
__device__ inline int 
quadratic(const QFLOAT b, const QFLOAT c, QFLOAT rts[4], const QFLOAT dis)
{
   int nquad=0;
   QFLOAT rtdis ;

   if (dis > doub0){
      nquad = 2 ;
      rtdis = sqrt(dis) ;
      if (b > doub0) 
		  rts[0] = ( -b - rtdis)*inv2 ;
      else 
		  rts[0] = ( -b + rtdis)*inv2 ;
      if (rts[0] == doub0) 
		  rts[1] =  -b ;
      else 
		  rts[1] = c/rts[0] ;
   }
   else{
      nquad = 0;
      rts[0] = 0 ;
      rts[1] = 0 ;
   }
   return(nquad);
}
*/

// ----------------------------------------------------------------------------
__device__ inline QFLOAT cubic(const QFLOAT p, const QFLOAT q, const QFLOAT r)
/* ====================================================
   find the lowest real root of the cubic - 
       x**3 + p*x**2 + q*x + r = 0 
   input parameters - 
     p,q,r - coeffs of cubic equation. 
   output- 
     cubic - a real root. 
   global constants -
     rt3 - sqrt(3) 
     inv3 - 1/3 
     doubmax - square root of largest number held by machine 
   method - 
     see D.E. Littlewood, "A University Algebra" pp.173 - 6 
     initial version: Charles Prineas   April 1981 
   called by  neumark, descartes, ferrari.
   calls  acos3 
========================================================*/
{
	QFLOAT po3,po3sq,qo3;
	QFLOAT uo3,u2o3,uo3sq4,uo3cu4 ;
	QFLOAT v,vsq,wsq ;
	QFLOAT m,mcube,n;
	QFLOAT muo3,s,scube,t,cosk,sinsqk ;
	QFLOAT root;
   
	m = doub0;
	if ((p > doubmax) || (p < -doubmax)) 
	   root = -p;
	else if ((q > doubmax) || (q < -doubmax)){
		if (q > doub0) 
			root = -r/q;
		else if (q < 0) 
			root = -sqrt(-q);
		else 
			root = 0;
	}
	else if ((r > doubmax)|| (r <  -doubmax)) 
		root =  -curoot(r) ;
	else{
		po3 = p*inv3 ;
		po3sq = po3*po3 ;
		if (po3sq > doubmax) 
			root =  -p ;
		else{
			//po3cu = po3*po3sq ;
			v = r + po3*(po3sq + po3sq - q) ;
			if ((v > doubmax) || (v < -doubmax)) 
				root = -p ;
			else{
				vsq = v*v ;
				qo3 = q*inv3 ;
				uo3 = qo3 - po3sq ;
				u2o3 = uo3 + uo3 ;
				if ((u2o3 > doubmax) || (u2o3 < -doubmax)){
				if (p == doub0){
					if (q > doub0) 
						root =  -r/q ;
					else if (q < doub0) 
						root =  -sqrt(-q);
					else root =  doub0 ;
				}
				else 
					root =  -q/p ;
            }
            uo3sq4 = u2o3*u2o3 ;
            if (uo3sq4 > doubmax){
				if (p == doub0){
					if (q > doub0) 
						root = -r/q ;
					else if (q < doub0) 
						root = -sqrt(-q) ;
					else 
						root = doub0;
				}
				else 
					root = -q/p ;
            }
            uo3cu4 = uo3sq4*uo3 ;
            wsq = uo3cu4 + vsq ;
            if (wsq > doub0){
			//cubic has one real root 
				if (v <= doub0) mcube = ( -v + sqrt(wsq))*inv2 ;
				if (v  > doub0) mcube = ( -v - sqrt(wsq))*inv2 ;
				m = curoot(mcube) ;
				if (m != doub0) 
					n = -uo3/m ;
				else 
					n = doub0;
				root = m + n - po3 ;
			}
            else{
				//cubic has three real roots 
				if (uo3 < doub0){
					muo3 = -uo3;
					if (muo3 > doub0) 
						s = sqrt(muo3);
					else 
						s = doub0;
					scube = s*muo3;
					if (scube == doub0){
						//exit(1);
					}
					t =  -v/(scube+scube) ;
					cosk = acos3(t) ;
					if (po3 < doub0)
						root = (s+s)*cosk - po3;
					else{
						sinsqk = doub1 - cosk*cosk ;
						if (sinsqk > doub0)
							root = s*( -cosk - rt3*sqrt(sinsqk)) - po3 ;
						else 
							root = -s*cosk - po3;
					}
				}
				else
					//cubic has multiple root -  
					root = curoot(v) - po3 ;
				}
			}
		}
   }
   return root;
}

// ----------------------------------------------------------------------------
/*
__device__ inline int 
descartes(const QFLOAT& a, const QFLOAT& b, const QFLOAT& c, const QFLOAT& d, QFLOAT rts[4])
//   Solve quartic equation using
//   Descartes-Euler-Cardano algorithm
//   Strong, T. "Elemementary and Higher Algebra"
//      Pratt and Oakley, p. 469 (1859)
//     29 Jun 1994  Don Herbison-Evans
{
   int nrts;
   int r1,r2;
   QFLOAT v1[4],v2[4];
   QFLOAT y;
   QFLOAT p,q,r;
   QFLOAT A,B,C;
   QFLOAT m,n1,n2;
   QFLOAT asq;
   QFLOAT Ainv4;
   QFLOAT Binvm;
   const QFLOAT d3o8 = 3.0/8.0;
   const QFLOAT inv8 = doub1/8.0;
   const QFLOAT inv16 = doub1/16.0;
   const QFLOAT d3o256 = 3.0/256.0;

   asq = a*a;
   A = b - asq*d3o8;
   B = c + a*(asq*inv8 - b*inv2);
   C = d + asq*(b*inv16 - asq*d3o256) - a*c*inv4;

   p = doub2*A;
   q = A*A - doub4*C;
   r = -B*B;

   y = cubic(p,q,r) ;
   if (y <= doub0) 
      nrts = 0;
   else{
      m = sqrt(y);
      Ainv4 = a*inv4;
      Binvm = B/m;
      n1 = (y + A + Binvm)*inv2;
      n2 = (y + A - Binvm)*inv2;
      r1 = quadratic(-m, n1, v1, y-doub4*n1);
      r2 = quadratic( m, n2, v2, y-doub4*n2);
      rts[0] = v1[0]-Ainv4;
      rts[1] = v1[1]-Ainv4;
      rts[r1] = v2[0]-Ainv4;
      rts[r1+1] = v2[1]-Ainv4;
      nrts = r1+r2;
   } 
   return(nrts);
}
*/

// ----------------------------------------------------------------------------
/*
__device__ inline int 
ferrari(const QFLOAT a, const QFLOAT b, const QFLOAT c, const QFLOAT d, QFLOAT rts[4])
//==================================================== 
// solve the quartic equation - 
// x**4 + a*x**3 + b*x**2 + c*x + d = 0 
// called by quartic
// calls cubic, quadratic.
//
// input - 
//		a,b,c,e - coeffs of equation. 
// output - 
//		nquar - number of real roots. 
//		rts - array of root values. 
// method :  Ferrari - Lagrange
//    Theory of Equations, H.W. Turnbull p. 140 (1947)
// calls  cubic, quadratic 
======================================================
{
	int nquar,n1,n2 ;
	QFLOAT v1[4],v2[4] ;
	QFLOAT e,f,ef ;
	QFLOAT g,gg,h,hh;
	const QFLOAT ainv2 = a*inv2;
	const QFLOAT asqinv4 = ainv2*ainv2;
	const QFLOAT d4 = d*doub4;
	const QFLOAT p = b ;
	const QFLOAT q = a*c-d4 ;
	const QFLOAT r = (asqinv4 - b)*d4 + c*c ;
	const QFLOAT y = cubic(p,q,r) ;
	const QFLOAT yinv2 = y*inv2;
	const QFLOAT esq = asqinv4 - b - y;
	const QFLOAT fsq = yinv2*yinv2 - d;

	if ((esq < doub0) && (fsq < doub0)) 
		nquar = 0;
	else{
		ef = -(inv4*a*y + inv2*c);
		if ( ((a > doub0)&&(y > doub0)&&(c > doub0))
			|| ((a > doub0)&&(y < doub0)&&(c < doub0))
			|| ((a < doub0)&&(y > doub0)&&(c < doub0))
			|| ((a < doub0)&&(y < doub0)&&(c > doub0))
			||  (a == doub0)||(y == doub0)||(c == doub0)){
			// use ef - 
            if ((b < doub0)&&(y < doub0)){
               e = sqrt(esq);
               f = ef/e;
            }
            else if (d < doub0){
               f = sqrt(fsq);
               e = ef/f;
            }
            else{
               if (esq > doub0) e = sqrt(esq); else e = doub0;
               if (fsq > doub0) f = sqrt(fsq); else f = doub0;
               if (ef < doub0) f = -f;
            }
        }
		else{// use esq and fsq - 
            if (esq > doub0) e = sqrt(esq); else e = doub0;
            if (fsq > doub0) f = sqrt(fsq); else f = doub0;
            if (ef < doub0) f = -f;
		}
		/// note that e >= doub0 
        g = ainv2 - e;
        gg = ainv2 + e;
        if (((b > doub0)&&(y > doub0)) || ((b < doub0)&&(y < doub0))){
            if ((a > doub0) && (e > doub0) || (a < doub0) && (e < doub0) ) 
				g = (b + y)/gg;
            else if ((a > doub0) && (e < doub0) || (a < doub0) && (e > doub0) ) 
				gg = (b + y)/g;
        }
        h = -yinv2 - f;
        hh = -yinv2 + f;
		if ( ((f > doub0)&&(y < doub0)) || ((f < doub0)&&(y > doub0)) ) 
			h = d/hh;
        else if ( ((f < doub0)&&(y < doub0)) || ((f > doub0)&&(y > doub0)) ) 
			hh = d/h;
        n1 = quadratic(gg, hh, v1, gg*gg-doub4*hh) ;
        n2 = quadratic(g, h, v2, g*g-doub4*h) ;
        nquar = n1+n2 ;
        rts[0] = v1[0] ;
        rts[1] = v1[1] ;
        rts[n1+0] = v2[0] ;
        rts[n1+1] = v2[1] ;
	}
	return nquar;
} 
*/

__device__ inline QuarticSolverResult 
copyResults(QuarticSolverResult result1, QuarticSolverResult result2)
{
	QuarticSolverResult result={0,{0,0,0,0}};
	const int n1 = result1.num;
	result.num = n1 + result2.num;
	result.roots[0] = result1.roots[0];
	result.roots[1] = result1.roots[1];
	result.roots[n1] = result2.roots[0];
	result.roots[n1+1] = result2.roots[1];
	return result;
}


__device__ inline QuarticSolverResult 
ferrari(const QFLOAT a, const QFLOAT b, const QFLOAT c, const QFLOAT d)
{
	QuarticSolverResult result={0,{0,0,0,0}};
	QFLOAT e,f,ef ;
	const QFLOAT ainv2 = a*inv2;
	const QFLOAT asqinv4 = ainv2*ainv2;
	const QFLOAT d4 = d*doub4;
	const QFLOAT p = b ;
	const QFLOAT q = a*c-d4 ;
	const QFLOAT r = (asqinv4 - b)*d4 + c*c ;
	const QFLOAT y = cubic(p,q,r) ;
	const QFLOAT yinv2 = y*inv2;
	const QFLOAT esq = asqinv4 - b - y;
	const QFLOAT fsq = yinv2*yinv2 - d;

	if ((esq < doub0) && (fsq < doub0)) 
		result.num = 0;
	else{
		ef = -(inv4*a*y + inv2*c);
		if ( ((a > doub0)&&(y > doub0)&&(c > doub0)) || ((a > doub0)&&(y < doub0)&&(c < doub0))
			 || ((a < doub0)&&(y > doub0)&&(c < doub0)) || ((a < doub0)&&(y < doub0)&&(c > doub0))
			 || (a == doub0) || (y == doub0) || (c == doub0) ){
			// use ef - 
            if ((b < doub0)&&(y < doub0)){
               e = sqrt(esq);
               f = ef/e;
            }
            else if (d < doub0){
               f = sqrt(fsq);
               e = ef/f;
            }
            else{
               if (esq > doub0) e = sqrt(esq); else e = doub0;
               if (fsq > doub0) f = sqrt(fsq); else f = doub0;
               if (ef < doub0) f = -f;
            }
        }
		else{// use esq and fsq -
            if (esq > doub0) e = sqrt(esq); else e = doub0;
            if (fsq > doub0) f = sqrt(fsq); else f = doub0;
            if (ef < doub0) f = -f;
		}
		// note that e >= doub0 
        QFLOAT g = ainv2 - e;
        QFLOAT gg = ainv2 + e;
        if (((b > doub0)&&(y > doub0)) || ((b < doub0)&&(y < doub0))){
            if ((a > doub0) && (e > doub0) || (a < doub0) && (e < doub0) ) 
				g = (b + y)/gg;
            else if ((a > doub0) && (e < doub0) || (a < doub0) && (e > doub0) ) 
				gg = (b + y)/g;
        }
        QFLOAT h = -yinv2 - f;
        QFLOAT hh = -yinv2 + f;
		if ( ((f > doub0)&&(y < doub0)) || ((f < doub0)&&(y > doub0)) ) 
			h = d/hh;
        else if ( ((f < doub0)&&(y < doub0)) || ((f > doub0)&&(y > doub0)) ) 
			hh = d/h;
		//=======================================
        QuarticSolverResult result1 = quadratic(gg, hh, gg*gg-doub4*hh) ;
        QuarticSolverResult result2 = quadratic(g, h, g*g-doub4*h);
		result = copyResults(result1, result2);
	}

	return result;
} // ferrari 


__device__ inline QuarticSolverResult 
neumark(const QFLOAT a, const QFLOAT b, const QFLOAT c, const QFLOAT d)
   //solve the quartic equation - 
   //x**4 + a*x**3 + b*x**2 + c*x + d = 0 
   //called by quartic
   //calls cubic, quadratic.
   //input parameters - 
   //	a,b,c,e - coeffs of equation. 
   //output parameters - 
   //	nquar - number of real roots. 
   //	rts - array of root values. 
   //method -  S. Neumark 
   //	Solution of Cubic and Quartic Equations - Pergamon 1965 
{
   QFLOAT g,gg,h,hh,gdis,gdisrt,hdis,hdisrt,g1,g2,h1,h2 ;
   QFLOAT bmy,gerr,herr,y4,bmysq ;
   QFLOAT hmax,gmax ;
   QuarticSolverResult result={0,{0,0,0,0}};

   QFLOAT asq = a*a ;
   QFLOAT d4 = d*doub4;
   QFLOAT p =  -b*doub2 ;
   QFLOAT q = b*b + a*c - d4 ;
   QFLOAT r = (c - a*b)*c + asq*d ;
   QFLOAT y = cubic(p,q,r) ;

   bmy = b - y ;
   y4 = y*doub4 ;
   bmysq = bmy*bmy ;
   gdis = asq - y4 ;
   hdis = bmysq - d4 ;
   if ((gdis <= doub0) || (hdis <= doub0))
	   result.num = 0;
   else{
      g1 = a*inv2 ;
      h1 = bmy*inv2 ;
      gerr = asq + y4 ;
      herr = hdis ;
      if (d > doub0) herr = bmysq + d4 ;
      if ((y < doub0) || (herr*gdis > gerr*hdis))
      {
         gdisrt = sqrt(gdis) ;
         g2 = gdisrt*inv2 ;
         if (gdisrt != doub0) h2 = (a*h1 - c)/gdisrt ;
            else h2 = doub0;
      }
      else
      {
         hdisrt = sqrt(hdis) ;
         h2 = hdisrt*inv2 ;
         if (hdisrt != doub0) g2 = (a*h1 - c)/hdisrt ;
            else g2 = doub0;
      }
     //note that in the following, the tests ensure non-zero denominators -  
      h = h1 - h2 ;
      hh = h1 + h2 ;
      hmax = hh ;
      if (hmax < doub0) hmax =  -hmax ;
      if (hmax < h) hmax = h ;
      if (hmax <  -h) hmax =  -h ;
      if ((h1 > doub0)&&(h2 > doub0)) h = d/hh ;
      if ((h1 < doub0)&&(h2 < doub0)) h = d/hh ;
      if ((h1 > doub0)&&(h2 < doub0)) hh = d/h ;
      if ((h1 < doub0)&&(h2 > doub0)) hh = d/h ;
      if (h > hmax) h = hmax ;
      if (h <  -hmax) h =  -hmax ;
      if (hh > hmax) hh = hmax ;
      if (hh <  -hmax) hh =  -hmax ;

      g = g1 - g2 ;
      gg = g1 + g2 ;
      gmax = gg ;
      if (gmax < doub0) gmax =  -gmax ;
      if (gmax < g) gmax = g ;
      if (gmax <  -g) gmax =  -g ;
      if ((g1 > doub0)&&(g2 > doub0)) g = y/gg ;
      if ((g1 < doub0)&&(g2 < doub0)) g = y/gg ;
      if ((g1 > doub0)&&(g2 < doub0)) gg = y/g ;
      if ((g1 < doub0)&&(g2 > doub0)) gg = y/g ;
      if (g > gmax) g = gmax ;
      if (g <  -gmax) g =  -gmax ;
      if (gg > gmax) gg = gmax ;
      if (gg <  -gmax) gg =  -gmax ;
 
      QuarticSolverResult result1 = quadratic(gg, hh, gg*gg - doub4*hh);
      QuarticSolverResult result2 = quadratic(g, h, g*g - doub4*h);
	  result = copyResults(result1, result2);
   }
   return result;
} 
//===neumark===


// ----------------------------------------------------------------------------
// Solve quartic equation using either
// quadratic, Ferrari's or Neumark's algorithm.
// called by
// calls  quadratic, ferrari, neumark.
// 21 Jan 1989  Don Herbison-Evans
__device__ inline QuarticSolverResult etkQuartic(const QFLOAT a, const QFLOAT b, const QFLOAT c, const QFLOAT d)
{
	QFLOAT odd, even;
	QFLOAT rts[4]={0,0,0,0};
	QuarticSolverResult r={0,{0,0,0,0}};

	if (a < doub0) odd = -a; else odd = a;
	if (c < doub0) odd -= c; else odd += c;
	if (b < doub0) even = -b; else even = b;
	if (d < doub0) even -= d; else even += d;

//First try bi-quadratic solver:
	if (odd > even*doubtol){
		int k;
		if (a < doub0) k  = 1;   else k = 0;
		if (b < doub0) k += k+1; else k +=k; 
		if (c < doub0) k += k+1; else k +=k; 
		if (d < doub0) k += k+1; else k +=k; 
		if (k==0 || k==3 || k==4 || k==6 || k==7 || k==9 || k==10 || (k>=12 && k<=15))
			r = ferrari(a,b,c,d);
		else
			r = neumark(a,b,c,d); 
	    if (r.num>0) 
			return r;
	}

//In case not success, we can still try the quadratic solver
	{
		r = quadratic(b, d, b*b - 4*d);
		QFLOAT roots[2]={r.roots[0], r.roots[1]};
		int j = 0;
		for (int k=0; k<r.num; k++){
			if (roots[k]>doub0){
				rts[j] = sqrt(roots[k]);
				rts[j+1] = -rts[j];
				++j; ++j;
			}
		}
		r.num = j;
		r.roots[0] = rts[0];
		r.roots[1] = rts[1];
		r.roots[2] = rts[2];
		r.roots[3] = rts[3];
	}
	return r;
}
