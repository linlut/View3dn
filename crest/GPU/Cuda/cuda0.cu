#include "hip/hip_runtime.h"
//CUDA interface file

#ifdef _WIN32
#  define WINDOWS_LEAN_AND_MEAN
#  define NOMINMAX
#  include <windows.h>
#endif

// includes, system
#include <stdlib.h>
#include <stdio.h>

// includes, GL
#include <GL/glew.h>
#include <GL/glut.h>

// includes
#include <hip/hip_runtime.h>
#include <cutil.h>
#include <cudaGL.h>
#include <cutil_math.h>
#include <cutil_gl_error.h>
#include <cuda_gl_interop.h>
#include <crest/system/ztime.h>
#include "cuda0.h"


void cudaCheckErrorGL(void)
{
    CUT_CHECK_ERROR_GL();
}

void gpuCudaGLRegisterBufferObject(GLuint vbo)
{
    // register buffer object with CUDA
    CUDA_SAFE_CALL(cudaGLRegisterBufferObject(vbo));    
}

void gpuCudaGLUnregisterBufferObject(GLuint vbo)
{
    CUDA_SAFE_CALL(cudaGLUnregisterBufferObject(vbo));
}

void cudaDeviceInit(void)
{
	CUT_DEVICE_INIT();	
	
	//Interoperability with OpenGL initialized
	cuGLInit();
}

void glewDeviceInit(void)
{
    glewInit();    
    if (!glewIsSupported( "GL_VERSION_2_0 " 
        "GL_ARB_pixel_buffer_object")) {
        fprintf( stderr, "ERROR: Support for necessary OpenGL extensions missing.\n");
        fflush( stderr);
    }	
    
    if (!glewIsSupported(
        "GL_VERSION_2_0 "
		"GL_ARB_vertex_program "
		"GL_ARB_fragment_program "
        "GL_ARB_texture_float "
		"GL_NV_gpu_program4 " // include GL_NV_geometry_program4
        "GL_ARB_texture_rectangle "
		))
    {
        printf("Unable to load extension()s:\n  GL_ARB_vertex_program\n  GL_ARB_fragment_program\n"
               "  GL_ARB_texture_float\n  GL_NV_gpu_program4\n  GL_ARB_texture_rectangle\n  OpenGL Version 2.0\nExiting...\n");
        exit(-1);
    }
}

void cudaDeviceClose(const int argc, char **argv)
{
    CUT_EXIT(argc, argv);
}

void cudaAllocateDeviceArray(void **devPtr, size_t size)
{
    CUDA_SAFE_CALL(hipMalloc(devPtr, size));
}

void cudaFreeDeviceArray(void *devPtr)
{
    CUDA_SAFE_CALL(hipFree(devPtr));
}

void cudaCopyArrayFromDevice(void* host, const void* device, unsigned int vbo, int size)
{   
    if (vbo)
        CUDA_SAFE_CALL(cudaGLMapBufferObject((void**)&device, vbo));
    CUDA_SAFE_CALL(hipMemcpy(host, device, size, hipMemcpyDeviceToHost));
    if (vbo)
        CUDA_SAFE_CALL(cudaGLUnmapBufferObject(vbo));
}

void cudaCopyArrayToDevice(void* device, const void* host, int offset, int size)
{
    CUDA_SAFE_CALL(hipMemcpy((char *) device + offset, host, size, hipMemcpyHostToDevice));
}


//======================GPU code=============================================

#define QFLOAT float
typedef struct
{
	float x[9];
} matrix3x3;
	
__device__ inline 
float3 mat_float3_mult(const matrix3x3 rot, const float3 t)
{
	float3 r;
	r.x = rot.x[0]*t.x + rot.x[1]*t.y + rot.x[2]*t.z;
	r.y = rot.x[3]*t.x + rot.x[4]*t.y + rot.x[5]*t.z;
	r.z = rot.x[6]*t.x + rot.x[7]*t.y + rot.x[8]*t.z;
	return r;
}

#include <crest/GPU/Cuda/etkQuaternion.cu>
#include <crest/GPU/Cuda/etkQuartic.cu>
#include <crest/GPU/Cuda/etkRegistration.cu>


__device__ inline float Distance2(float3 p, float3 q)
{
	float3 r = p - q;
	float o = r.x*r.x+r.y*r.y+r.z*r.z;
	return o;
}

__device__ inline 
void computeSquareDistanceWeights(const float3& center, const float3 refVertex[4], QFLOAT weights[4])
{
	QFLOAT dist0 = Distance2(center, refVertex[0]);
	QFLOAT dist1 = Distance2(center, refVertex[1]);
	QFLOAT dist2 = Distance2(center, refVertex[2]);
	QFLOAT dist3 = Distance2(center, refVertex[3]);
	QFLOAT bias = (dist0+dist1+dist2+dist3)*0.25*0.03;
	weights[0] = 1/(dist0+bias);
	weights[1] = 1/(dist1+bias);
	weights[2] = 1/(dist2+bias);
	weights[3] = 1/(dist3+bias);
	QFLOAT sum_w= weights[0]+weights[1]+weights[2]+weights[3];
	QFLOAT inv_sum_w = 1.0/sum_w;
	weights[0] *= inv_sum_w;
	weights[1] *= inv_sum_w;
	weights[2] *= inv_sum_w;
	weights[3] *= inv_sum_w;
}

///////////////////////////////////////////////////////////////////////////////
//! @param data  data in global memory
///////////////////////////////////////////////////////////////////////////////
__global__ 
void gpuMlsDeformation(
	const unsigned int width,//Vertex ARRAY WIDTH
	const float3 *refVert,	 //reference vertex array in their initial positions
	const float3 *defRefVert,//reference vertex array-deformed 
	const float3 *vert,		 //the vertex array to be deformed	
	const int4 *conn,		 //index to the references
	float3 *defv,			 //deformed vertices
	float4 *quat)			 //rotation matrix in deformation
{
	float3 u, p[4], q[4];
	float weights[4];
    const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
	const unsigned int index = y*width+x;	

	int4 refVertID = conn[index];
	u = vert[index];	
		
	//setup static vertices	
	p[0] = refVert[refVertID.x];
	p[1] = refVert[refVertID.y];
	p[2] = refVert[refVertID.z];
	p[3] = refVert[refVertID.w];
		
	//get wieghts
	computeSquareDistanceWeights(u, p, weights);
	
	//get reference points
	q[0] = defRefVert[refVertID.x];
	q[1] = defRefVert[refVertID.y];
	q[2] = defRefVert[refVertID.z];
	q[3] = defRefVert[refVertID.w];

	//computation
	const int nPoints=4;
	float4 result_quat;
	matrix3x3 result_rot;
	float3 result_trans;
	_etkRegister(nPoints, p, q, weights, &result_quat, &result_rot, &result_trans);

    //write output vertex 
    quat[index] = result_quat;    
    defv[index] = result_trans + mat_float3_mult(result_rot, u);
}


//===============================CPU again==============================
void runGpuMlsDeformation(
	const int nv,					//length of the problem (here is the vertex length)
	GLuint vboRefVertexArray,		//static vertex array of the reference model, float3
	GLuint vboDefRefVertexArray,    //deformed vertex array of the reference model, float3
	//float *pDeviceDefRefVertex,
	GLuint vboNeighborArray,		//neighbourhood array, or connectivity, int4
	GLuint vboVertexArray,		    //static vertex array of the visual model, float3
	GLuint vboDefVertexArray,		//deformed vertex array of the visual model, float3
	GLuint vboQuatArray)		    //rotation quaternion array, float4
{
	const unsigned int shiftbit = ARRAY_LEN_SHIFTBIT;
	const unsigned int padding = ARRAY_LEN_PADDING;
	const unsigned int mesh_width=16;
	const unsigned int mesh_height=(((nv+padding)>>shiftbit)<<shiftbit)/mesh_width;
	
    // map OpenGL buffer objects for writing from CUDA
	float3 *refVert=NULL;	//reference vertex array in their initial positions
	float3 *defRefVert=NULL;//reference vertex array-deformed 
	int4 *conn=NULL;		//index to the references
	float3 *vert=NULL;		//the vertex array to be deformed	
	float3 *defv=NULL;		//deformed vertices
	float4 *quat=NULL;		//rotation matrix in deformation	
    CUDA_SAFE_CALL(cudaGLMapBufferObject((void**)&refVert, vboRefVertexArray));
    CUDA_SAFE_CALL(cudaGLMapBufferObject((void**)&defRefVert, vboDefRefVertexArray));
    //defRefVert = (float3*)pDeviceDefRefVertex;
    CUDA_SAFE_CALL(cudaGLMapBufferObject((void**)&conn, vboNeighborArray));
    CUDA_SAFE_CALL(cudaGLMapBufferObject((void**)&vert, vboVertexArray));    
    CUDA_SAFE_CALL(cudaGLMapBufferObject((void**)&defv, vboDefVertexArray));
    CUDA_SAFE_CALL(cudaGLMapBufferObject((void**)&quat, vboQuatArray));

    // execute the kernel
    dim3 block(16, 8, 1);
    dim3 grid(mesh_width/block.x, mesh_height/block.y, 1);
    gpuMlsDeformation<<<grid, block>>>(mesh_width, refVert, defRefVert, vert, conn, defv, quat);

    // unmap buffer objects
    CUDA_SAFE_CALL(cudaGLUnmapBufferObject( vboQuatArray));
    CUDA_SAFE_CALL(cudaGLUnmapBufferObject( vboDefVertexArray));
    CUDA_SAFE_CALL(cudaGLUnmapBufferObject( vboVertexArray));
    CUDA_SAFE_CALL(cudaGLUnmapBufferObject( vboNeighborArray));
    CUDA_SAFE_CALL(cudaGLUnmapBufferObject( vboDefRefVertexArray)); 
    CUDA_SAFE_CALL(cudaGLUnmapBufferObject( vboRefVertexArray));
}

/*
CUT_SAFE_CALL(cutResetTimer(hTimer));
CUT_SAFE_CALL(cutStartTimer(hTimer));

VectorAddition<<< grid, threads >>>( vector1_gpu, vector2_gpu, result_gpu);
CUDA_SAFE_CALL(hipDeviceSynchronize());

performanceMetrics.exe_GPU_time += cutGetTimerValue(hTimer);
*/
