#include "hip/hip_runtime.h"
//---------------------------------------------------------------------------
// PRIVATE: determinant of a 3x3 matrix
// ----------------------------------------------------------------------------

__device__ inline 
QFLOAT etkDet33 (const QFLOAT a1, const QFLOAT b1, const QFLOAT c1,
                 const QFLOAT a2, const QFLOAT b2, const QFLOAT c2,
                 const QFLOAT a3, const QFLOAT b3, const QFLOAT c3)
{
	QFLOAT r= a1*b2*c3 + b1*c2*a3 + c1*a2*b3 - a3*b2*c1 - b3*c2*a1 - c3*a2*b1;
	return r;
}

//---------------------------------------------------------------------------
// PRIVATE: comatrix of a 4x4 matrix
// ----------------------------------------------------------------------------
struct mat4x4{
	QFLOAT a1, b1, c1, d1;
	QFLOAT a2, b2, c2, d2;
	QFLOAT a3, b3, c3, d3;
	QFLOAT a4, b4, c4, d4;
};

__device__ inline mat4x4
etkComatrix44 (QFLOAT a1, QFLOAT b1, QFLOAT c1, QFLOAT d1,
               QFLOAT a2, QFLOAT b2, QFLOAT c2, QFLOAT d2,
               QFLOAT a3, QFLOAT b3, QFLOAT c3, QFLOAT d3,
               QFLOAT a4, QFLOAT b4, QFLOAT c4, QFLOAT d4)
{
	QFLOAT coffa1, coffa2, coffa3, coffa4, coffb1, coffb2, coffb3, coffb4,
		   coffc1, coffc2, coffc3, coffc4, coffd1, coffd2, coffd3, coffd4;
	mat4x4 m;

	coffa1 =  etkDet33(b2, c2, d2, b3, c3, d3, b4, c4, d4);
	coffb1 = -etkDet33(a2, c2, d2, a3, c3, d3, a4, c4, d4);
	coffc1 =  etkDet33(a2, b2, d2, a3, b3, d3, a4, b4, d4);
	coffd1 = -etkDet33(a2, b2, c2, a3, b3, c3, a4, b4, c4);
	coffa2 = -etkDet33(b1, c1, d1, b3, c3, d3, b4, c4, d4);
	coffb2 =  etkDet33(a1, c1, d1, a3, c3, d3, a4, c4, d4);
	coffc2 = -etkDet33(a1, b1, d1,a3, b3, d3, a4, b4, d4);
	coffd2 =  etkDet33(a1, b1, c1,a3, b3, c3, a4, b4, c4);
	coffa3 =  etkDet33(b1, c1, d1,b2, c2, d2, b4, c4, d4);
	coffb3 = -etkDet33(a1, c1, d1,a2, c2, d2, a4, c4, d4);
	coffc3 =  etkDet33(a1, b1, d1,a2, b2, d2, a4, b4, d4);
	coffd3 = -etkDet33(a1, b1, c1,a2, b2, c2, a4, b4, c4);
	coffa4 = -etkDet33(b1, c1, d1, b2, c2, d2, b3, c3, d3);
	coffb4 =  etkDet33(a1, c1, d1, a2, c2, d2, a3, c3, d3);
	coffc4 = -etkDet33(a1, b1, d1, a2, b2, d2, a3, b3, d3);
	coffd4 =  etkDet33(a1, b1, c1, a2, b2, c2, a3, b3, c3);
	m.a1 = coffa1;  m.a2 = coffa2;  m.a3 = coffa3;  m.a4 = coffa4;
	m.b1 = coffb1;  m.b2 = coffb2;  m.b3 = coffb3;  m.b4 = coffb4;
	m.c1 = coffc1;  m.c2 = coffc2;  m.c3 = coffc3;  m.c4 = coffc4;
	m.d1 = coffd1;  m.d2 = coffd2;  m.d3 = coffd3;  m.d4 = coffd4;
	return m;
}

// ----------------------------------------------------------------------------
// PRIVATE: eigen value decomposition of a 4x4 matrix
// ----------------------------------------------------------------------------
__device__ inline float4 
etkRegisterMatrix(
			QFLOAT dN11, QFLOAT dN12, QFLOAT dN13, QFLOAT dN14,
			QFLOAT dN21, QFLOAT dN22, QFLOAT dN23, QFLOAT dN24,
			QFLOAT dN31, QFLOAT dN32, QFLOAT dN33, QFLOAT dN34,
			QFLOAT dN41, QFLOAT dN42, QFLOAT dN43, QFLOAT dN44)
{
	// Determinant of N - y*I
	QFLOAT dByyy, dCyy, dDy, dE;
	int i;

	// Ay^4 + By^3 + Cy^2 + Dy + E = 0;
	// ---- First row of the determinant ----
	// 1.1
	dByyy = - dN11 - dN22 - dN33 - dN44;
	dCyy  =   dN11*dN22 + dN11*dN33 + dN11*dN44 + dN22*dN33 + dN22*dN44 + dN33*dN44;
	dDy	  = - dN11*dN22*dN33 - dN11*dN22*dN44 - dN11*dN33*dN44 - dN22*dN33*dN44;
	dE	  =   dN11*dN22*dN33*dN44;
	// 1.2
	dDy  -=	dN23*dN34*dN42;
	dE   +=	dN11*dN23*dN34*dN42;
	// 1.3
	dDy  -=	dN24*dN32*dN43;
	dE   +=	dN11*dN24*dN32*dN43;
	// 1.4
	dCyy -=   dN34*dN43;
	dDy  +=	(dN11 + dN22)*dN34*dN43;
	dE   -=	dN11*dN22*dN34*dN43;
	// 1.5
	dCyy -=	dN23*dN32;
	dDy  +=	(dN11 + dN44)*dN23*dN32;
	dE   -=	dN11*dN23*dN32*dN44;
	// 1.6
	dCyy -=	dN24*dN42;
	dDy  +=	(dN11 + dN33)*dN24*dN42;
	dE   -=	dN11*dN24*dN33*dN42;

	// ---- Second row of the determinant ----
	// 2.1
	dCyy -= dN21*dN12;
	dDy  += dN21*dN12*(dN33 + dN44);
	dE   -= dN21*dN12*dN33*dN44;
	// 2.2
	dE  -= dN21*dN13*dN34*dN42;
	// 2.3
	dE  -= dN21*dN14*dN32*dN43;
	// 2.4
	dE += dN21*dN12*dN34*dN43;
	// 2.5
	dDy -= dN21*dN13*dN32;
	dE += dN21*dN13*dN32*dN44;
	// 2.6
	dDy -= dN21*dN14*dN42;
	dE  += dN21*dN14*dN33*dN42;

	// ---- Third row of the determinant ----
	// 3.1
	dDy -= dN31*dN12*dN23;
	dE += dN31*dN12*dN23*dN44;
	// 3.2
	dE += dN31*dN13*dN24*dN42;
	// 3.3
	dDy -= dN31*dN14*dN43;
	dE += dN31*dN14*dN22*dN43;
	// 3.4
	dE -= dN31*dN12*dN24*dN43;
	// 3.5
	dCyy -= dN31*dN13;
	dDy += dN31*dN13*(dN22 + dN44);
	dE -= dN31*dN13*dN22*dN44;
	// 3.6
	dE -= dN31*dN14*dN23*dN42;

	// ---- Forth row of the determinant ----
	// 4.1
	dE -= dN41*dN12*dN23*dN34;
	// 4.2
	dE -= dN41*dN13*dN24*dN32;
	// 4.3
	dCyy -= dN41*dN14;
	dDy += dN41*dN14*(dN22 + dN33);
	dE -= dN41*dN14*dN22*dN33;
	// 4.4
	dDy -= dN41*dN12*dN24;
	dE += dN41*dN12*dN24*dN33;
	// 4.5
	dDy -= dN41*dN13*dN34;
	dE += dN41*dN13*dN34*dN22;
	// 4.6
	dE += dN41*dN14*dN23*dN32;

	//Solving the quartic equation to obtain the eigenvalues
	/*{
	QFLOAT Amax = fabs(dByyy);
	Amax = max(Amax, fabs(dCyy));
	Amax = max(Amax, fabs(dDy));
	Amax = max(Amax, fabs(dE));
	QFLOAT Amax1 = 1.0/Amax;
	dByyy *= Amax1;
	dCyy *= Amax1;
	dDy *= Amax1;
	dE *= Amax1;
	const QFLOAT TOL=1e-20;
	if (fabs(dByyy)<TOL) dByyy = 0;
	if (fabs(dCyy)<TOL) dCyy = 0;
	if (fabs(dDy)<TOL) dDy = 0;
	if (fabs(dE)<TOL) dE = 0;
	}*/
	QuarticSolverResult result = etkQuartic(dByyy, dCyy, dDy, dE);
	const int iNbRoots = result.num;
	const QFLOAT adRoots[4]={result.roots[0], result.roots[1], result.roots[2], result.roots[3]};

	// Finding the  argest eigenvalue
#ifdef __HIPCC__
	QFLOAT dMaxEigenValue = max(adRoots[0], adRoots[1]);
	dMaxEigenValue = max(dMaxEigenValue, adRoots[2]);
	dMaxEigenValue = max(dMaxEigenValue, adRoots[3]);
#else
	QFLOAT dMaxEigenValue = adRoots[0];
	for (i=1; i<iNbRoots; i++){
		if (adRoots[i] > dMaxEigenValue)
			dMaxEigenValue = adRoots[i];
	}
#endif

	QFLOAT dN11bis = dN11 - dMaxEigenValue;
	QFLOAT dN22bis = dN22 - dMaxEigenValue;
	QFLOAT dN33bis = dN33 - dMaxEigenValue;
	QFLOAT dN44bis = dN44 - dMaxEigenValue;
	mat4x4 m = etkComatrix44(
					dN11bis, dN12, dN13, dN14,
					dN21, dN22bis, dN23, dN24,
					dN31, dN32, dN33bis, dN34,
					dN41, dN42, dN43, dN44bis);
	dN11bis=m.a1, dN12=m.b1, dN13=m.c1, dN14=m.d1;
	dN21=m.a2, dN22bis=m.b2, dN23=m.c2, dN24=m.d2;
	dN31=m.a3, dN32=m.b3, dN33bis=m.c3, dN34=m.d3;
	dN41=m.a4, dN42=m.b4, dN43=m.c4, dN44bis=m.d4;
	QFLOAT qx=dN12+dN22bis+dN32+dN42;
	QFLOAT qy=dN13+dN23+dN33bis+dN43;
	QFLOAT qz=dN14+dN24+dN34+dN44bis;
	QFLOAT qw=dN11bis+dN21+dN31+dN41;	
	float4 quat = make_float4(qx, qy, qz, qw);
	return normalize4(quat);
}


// ----------------------------------------------------------------------------
// Register the two point sets.
// - Rotation matrix is stored in p33Rot,
// - Translation vector is stored in pTrans
// - Return error matrix
// ----------------------------------------------------------------------------

__device__ inline float3 weightedSum(const float3 p[4], const QFLOAT w[4], const int n)
{
	float3 sum = p[0]*w[0];
   #ifdef __HIPCC__
    sum+= p[1]*w[1];
    sum+= p[2]*w[2];
    sum+= p[3]*w[3];
   #else	
	for (int i=1; i<n; i++)
		sum += p[i]*w[i];
   #endif
	return sum;
}


__device__ inline void _etkRegister(
	const int nPoints, const float3 _adModelPoints[4], const float3 _adSensorPoints[4], const QFLOAT weights[4], //Input parameters
    float4 result_quat[1], matrix3x3 result_rot[1], float3 result_trans[1])  //return parameters
{
	int u;
	float3 adModelPoints[16];
	float3 adSensorPoints[16];

	// 1. Find centroid
	const float3 adModelCentroid = weightedSum(_adModelPoints, weights, nPoints);
	const float3 adSensorCentroid = weightedSum(_adSensorPoints, weights, nPoints);

	// 2. Recenter points;
   #ifdef __HIPCC__
        u = 0;
		adModelPoints[u]=_adModelPoints[u]-adModelCentroid;
		adSensorPoints[u]=_adSensorPoints[u]-adSensorCentroid;
        u = 1;
		adModelPoints[u]=_adModelPoints[u]-adModelCentroid;
		adSensorPoints[u]=_adSensorPoints[u]-adSensorCentroid;
        u = 2;
		adModelPoints[u]=_adModelPoints[u]-adModelCentroid;
		adSensorPoints[u]=_adSensorPoints[u]-adSensorCentroid;
        u = 3;
		adModelPoints[u]=_adModelPoints[u]-adModelCentroid;
		adSensorPoints[u]=_adSensorPoints[u]-adSensorCentroid;	
   #else
	for (u = 0; u < nPoints; u++){
		adModelPoints[u]=_adModelPoints[u]-adModelCentroid;
		adSensorPoints[u]=_adSensorPoints[u]-adSensorCentroid;
	}	   
   #endif
   	
	// 3. Compute momentum
	QFLOAT dSxx, dSxy, dSxz, dSyx, dSyy, dSyz, dSzx, dSzy, dSzz;
	dSxx=dSxy=dSxz=dSyx=dSyy=dSyz=dSzx=dSzy=dSzz=0;
	for (u = 0; u < nPoints; u++){
		const float3 v1 = adModelPoints[u];
		const float3 v2 = adSensorPoints[u];
		const QFLOAT w = weights[u];
		const QFLOAT p0=v1.x, p1=v1.y, p2=v1.z;
		const QFLOAT q0=v2.x*w, q1=v2.y*w, q2=v2.z*w;	
		dSxx += p0 * q0; dSxy += p0 * q1;
		dSxz += p0 * q2; dSyx += p1 * q0;
		dSyy += p1 * q1; dSyz += p1 * q2;
		dSzx += p2 * q0; dSzy += p2 * q1;
		dSzz += p2 * q2;
	}

	// N Symmetric Matrix
	QFLOAT dN11 = dSxx + dSyy + dSzz;
	QFLOAT dN12 = dSyz - dSzy;
	QFLOAT dN13 = dSzx - dSxz;
	QFLOAT dN14 = dSxy - dSyx;
	QFLOAT dN21 = dN12;
	QFLOAT dN22 = dSxx - dSyy - dSzz;
	QFLOAT dN23 = dSxy + dSyx;
	QFLOAT dN24 = dSzx + dSxz;
	QFLOAT dN31 = dN13;
	QFLOAT dN32 = dN23;
	QFLOAT dN33 = -dSxx + dSyy - dSzz;
	QFLOAT dN34 = dSyz + dSzy;
	QFLOAT dN41 = dN14;
	QFLOAT dN42 = dN24;
	QFLOAT dN43 = dN34;
	QFLOAT dN44 = -dSxx - dSyy + dSzz;

	//compute rotation
	result_quat[0] = etkRegisterMatrix(dN11, dN12, dN13, dN14, dN21, dN22, dN23, dN24, dN31, dN32, dN33, dN34, dN41, dN42, dN43, dN44);		
	
	//vector is column vector to mult the matrix, therefore, we use y=Ax
	result_rot[0] = quaterionToMatrix33_v2(result_quat[0]);	
	
    //compute translation
  #ifdef __HIPCC__
    //GPU implementation
	result_trans[0]= adSensorCentroid - mat_float3_mult(result_rot[0], adModelCentroid);
  #else 
  #ifdef __cplusplus
    //C++ CPU implementation
	result_trans[0] = adSensorCentroid - result_rot[0] * adModelCentroid;
  #else 
	//Cg language
	result_trans[0] = adSensorCentroid - mul(result_rot[0], adModelCentroid);
  #endif
  #endif
}


__device__ inline void EtkRegister(
	const int nPoints, 
	const float3 _adModelPoints[], 
	const float3 _adSensorPoints[], 
	const QFLOAT weights[],				//Input parameters
    float4 result_quat[1])				//return parameters
{
	const float3* adModelPoints = _adModelPoints;
	const float3* adSensorPoints = _adSensorPoints;
   	
	// 3. Compute momentum
	QFLOAT dSxx, dSxy, dSxz, dSyx, dSyy, dSyz, dSzx, dSzy, dSzz;
	dSxx=dSxy=dSxz=dSyx=dSyy=dSyz=dSzx=dSzy=dSzz=0;
	for (int u = 0; u < nPoints; u++){
		const float3& v1 = adModelPoints[u];
		const float3& v2 = adSensorPoints[u];
		const QFLOAT& w = weights[u];
		const QFLOAT p0=v1.x, p1=v1.y, p2=v1.z;
		const QFLOAT q0=v2.x*w, q1=v2.y*w, q2=v2.z*w;	
		dSxx += p0 * q0; dSxy += p0 * q1;
		dSxz += p0 * q2; dSyx += p1 * q0;
		dSyy += p1 * q1; dSyz += p1 * q2;
		dSzx += p2 * q0; dSzy += p2 * q1;
		dSzz += p2 * q2;
	}

	// N Symmetric Matrix
	QFLOAT dN11 = dSxx + dSyy + dSzz;
	QFLOAT dN12 = dSyz - dSzy;
	QFLOAT dN13 = dSzx - dSxz;
	QFLOAT dN14 = dSxy - dSyx;
	QFLOAT dN21 = dN12;
	QFLOAT dN22 = dSxx - dSyy - dSzz;
	QFLOAT dN23 = dSxy + dSyx;
	QFLOAT dN24 = dSzx + dSxz;
	QFLOAT dN31 = dN13;
	QFLOAT dN32 = dN23;
	QFLOAT dN33 = -dSxx + dSyy - dSzz;
	QFLOAT dN34 = dSyz + dSzy;
	QFLOAT dN41 = dN14;
	QFLOAT dN42 = dN24;
	QFLOAT dN43 = dN34;
	QFLOAT dN44 = -dSxx - dSyy + dSzz;

	//compute rotation
	result_quat[0] = etkRegisterMatrix(dN11, dN12, dN13, dN14, dN21, dN22, dN23, dN24, dN31, dN32, dN33, dN34, dN41, dN42, dN43, dN44);	
}


