#include "hip/hip_runtime.h"
//FILE: etkQuaternion.cu  
// I try to make this part compatible with NVIDIA CUDA and NVIDIA CG

/// Normalize a quaternion
__device__ inline float4 normalize4(const float4 quat)
{
	QFLOAT x = quat.x;
	QFLOAT y = quat.y;
	QFLOAT z = quat.z;
	QFLOAT w = quat.w;
	QFLOAT dNorm = sqrt(w*w + x*x + y*y + z*z);
	if (dNorm == 0){
		x = y = z = 0, w=1;
	}
	else{
		QFLOAT n1 = 1.0/dNorm;
		w *= n1;
		x *= n1;
		y *= n1;
		z *= n1;
	}
	//Avoid zero norm
	return make_float4(x,y,z,w);
}


/// Transform a quaternion to a 3x3 (rotation) matrix
__device__ inline matrix3x3 quaterionToMatrix33(const float4 quat)
{
	matrix3x3 mat33;
	const QFLOAT dX = quat.x;
	const QFLOAT dY = quat.y;
	const QFLOAT dZ = quat.z;
	const QFLOAT dW = quat.w;
	const QFLOAT dX2 = dX + dX;
	const QFLOAT dY2 = dY + dY;
	const QFLOAT dZ2 = dZ + dZ;
	const QFLOAT dXX = dX * dX2,
				 dXY = dX * dY2,
				 dXZ = dX * dZ2,
				 dYY = dY * dY2,
				 dYZ = dY * dZ2,
				 dZZ = dZ * dZ2,
				 dWX = dW * dX2,
				 dWY = dW * dY2,
				 dWZ = dW * dZ2;
	mat33.x[0] = 1 - (dYY + dZZ);
	mat33.x[1] = dXY + dWZ;
	mat33.x[2] = dXZ - dWY;
	mat33.x[3] = dXY - dWZ;
	mat33.x[4] = 1 - (dXX + dZZ);
	mat33.x[5] = dYZ + dWX;
	mat33.x[6] = dXZ + dWY;
	mat33.x[7] = dYZ - dWX;
	mat33.x[8] = 1 - (dXX + dYY);
	return mat33;
 }


__device__ inline matrix3x3 quaterionToMatrix33_v2(const float4 pquat)
{
	matrix3x3 mat33;
	const QFLOAT dX = pquat.x;
	const QFLOAT dY = pquat.y;
	const QFLOAT dZ = pquat.z;
	const QFLOAT dW = pquat.w;

	const QFLOAT dX2 = dX + dX;
	const QFLOAT dY2 = dY + dY;
	const QFLOAT dZ2 = dZ + dZ;
	const QFLOAT dXX = dX * dX2,
				 dXY = dX * dY2,
				 dXZ = dX * dZ2,
				 dYY = dY * dY2,
				 dYZ = dY * dZ2,
				 dZZ = dZ * dZ2,
				 dWX = dW * dX2,
				 dWY = dW * dY2,
				 dWZ = dW * dZ2;

	mat33.x[0] = 1 - (dYY + dZZ);
	mat33.x[3] = dXY + dWZ;
	mat33.x[6] = dXZ - dWY;
	mat33.x[1] = dXY - dWZ;
	mat33.x[4] = 1 - (dXX + dZZ);
	mat33.x[7] = dYZ + dWX;
	mat33.x[2] = dXZ + dWY;
	mat33.x[5] = dYZ - dWX;
	mat33.x[8] = 1 - (dXX + dYY);
	return mat33;
  }
